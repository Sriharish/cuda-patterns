﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void addArrays(int* a, int* b, int* c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main() {
	const int count = 5;
	int dataSize = count * sizeof(int);
	int hA[] = { 1, 2, 3, 4, 5 };
	int hB[] = { 10, 20, 30, 40, 50 };
	int hC[count];

	// Cuda Constants
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	hipDeviceProp_t prop;

	for (int i = 0; i < deviceCount; i++) {
		hipGetDeviceProperties(&prop, i);
		cout << "Device " << i << ": " << prop.name << endl;
		cout << "Compute Power: " << prop.major << "." << prop.minor << endl;
		cout << "Max grid dimensions: (" << prop.maxGridSize[0] << " x " <<
			prop.maxGridSize[1] << " x " <<
			prop.maxGridSize[2] << ")" << endl;
		cout << "Max block dimensions: (" << prop.maxThreadsDim[0] << " x " <<
			prop.maxThreadsDim[1] << " x " <<
			prop.maxThreadsDim[2] << ")" << endl << endl;
	}

	getchar();

	int* dA, *dB, *dC;
	hipMalloc(&dA, dataSize);
	hipMalloc(&dB, dataSize);
	hipMalloc(&dC, dataSize);
	
	hipMemcpy(dA, hA, dataSize, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, dataSize, hipMemcpyHostToDevice);
	
	addArrays <<<1, count>>>(dA, dB, dC);

	hipMemcpy(hC, dC, dataSize, hipMemcpyDeviceToHost);
	
	/*for (int i = 0; i < count; ++i) {
		addArrays(a, b, c);
	}*/
	
	for (int i = 0; i < count; ++i) {
		printf("%d ", hC[i]);
	}

	return 0;
}

