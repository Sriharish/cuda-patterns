﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void sum(int* dA) {
	int threadId = threadIdx.x;


}

int main() {

	const int count = 512;
	int computeSize = count * sizeof(int);
	int hA[count];
	int* dA;

	for (int i = 0; i < count; i++) {
		hA[i] = i + 1;
	}

	hipMalloc(&dA, computeSize);
	hipMemcpy(dA, hA, computeSize, hipMemcpyHostToDevice);

	sum<<<1, count - 1>>>(dA);

	hipMemcpy(hA, dA, computeSize, hipMemcpyDeviceToHost);
	hipFree(dA);

	cout << hA[count - 1] << endl;
	getchar();
	return 0;
}