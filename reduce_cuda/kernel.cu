﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void reduce(int* dA) {
	int threadId = threadIdx.x;
	
	for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1) {
		if (threadId < tc) {
			int pA = threadId * stepSize * 2;
			int pB = pA + stepSize;
			dA[pA] += dA[pB];
		}
	}
}

__global__ void reduce_shared(int* dA) {
	extern __shared__ int dACopy[];
	int threadId = threadIdx.x;
	dACopy[threadId * 2 + 1] = dA[threadId * 2];

	for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1) {
		if (threadId < tc) {
			int pA = threadId * stepSize * 2;
			int pB = pA + stepSize;
			dACopy[pA] += dACopy[pB];
		}
	}

	if (threadId == 0) {
		dA[0] = dACopy[0];
	}
}

int main() {
	const int count = 4;
	int computeSize = count * sizeof(int);
	int hA[] = {2, 2, 3, 4};
	int* dA;

	hipMalloc(&dA, computeSize);
	hipMemcpy(dA, hA, computeSize, hipMemcpyHostToDevice);
	
	int numThreads = count / 2;
	reduce_shared<<<1, numThreads, computeSize>>>(dA);
	
	hipMemcpy(hA, dA, computeSize, hipMemcpyDeviceToHost);
	hipFree(dA);

	for (int i = 0; i < count; ++i) {
		cout << hA[i] << endl;
	}

	getchar();
	return 0;
}