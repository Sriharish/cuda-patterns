﻿
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h";
#include <ctime>
#include <cstdio>;
#include <iostream>;

using namespace std;

__global__ void addTen(float* d, int count) {
	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;

	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;
	if (tid < count) {
		d[tid] = d[tid] * 1000 + 1;
	}
}


int main() {
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32); // init generator
	hiprandSetPseudoRandomGeneratorSeed(gen, time(0)); // provide seed value

	const int count = 123456;
	const int size = count * sizeof(float);
	float* d;
	float h[count];
	hipMalloc(&d, size);
	hiprandGenerateUniform(gen, d, count);
	
	dim3 block(8, 8, 8);
	dim3 grid(16, 16);

	addTen<<<grid, block>>>(d, count);
	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
	hipFree(d);

	for (int i = 0; i < count; ++i) {
		cout << h[i] << endl;
	}

	getchar();
	return 0;
}
